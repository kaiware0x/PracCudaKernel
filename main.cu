
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void minimal(int *d_a)
{
    *d_a = 13;
}

__global__ void assign(int *d_a, int value)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    d_a[idx] = value;
}

__global__ void assign2D(int *d_a, int w, int h, int value)
{
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = iy * w + ix;
    d_a[idx] = value;
}

void inc_cpu(int *a, int N)
{
    for (size_t idx = 0; idx < N; idx++)
    {
        a[idx]++;
    }
}

__global__ void inc_gpu(int *d_a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        d_a[idx]++;
    }
}
__device__ void device_strcpy(char *dst, const char *src)
{
    while (*dst++ = *src++)
        ;
}

__global__ void kernel(char *A)
{
    device_strcpy(A, "Hello, World!");
}

int main()
{
    char *d_hello;

    hipMalloc((void **)&d_hello, 32);
    
    kernel<<<1, 1>>>(d_hello);

    char hello[32];
    hipMemcpy(hello, d_hello, 32, hipMemcpyDeviceToHost);

    hipFree(d_hello);

    puts(hello);
}